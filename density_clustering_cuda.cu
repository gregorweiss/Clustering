#include "hip/hip_runtime.h"

#include "tools.hpp"
#include "density_clustering_cuda.hpp"

#include <algorithm>

#include <hip/hip_runtime.h>
#include <omp.h>

#include "lts_cuda_kernels.cuh"


#define BSIZE 128
#define N_STREAMS 1

namespace Clustering {
namespace Density {
namespace CUDA {

  void check_error() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
      exit(EXIT_FAILURE);
    }
  }

  __global__ void
  in_radius(unsigned int offset
          , float* sorted_coords
          , float* coords
          , unsigned int i_ref
          , unsigned int n_rows
          , unsigned int n_cols
          , float* radii2
          , unsigned int n_radii
          , float* in_radius) {
    //TODO store ref in local group?
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int gid = bid*BSIZE+tid;
    float c;
    float dist2 = 0.0f;
    unsigned int j,r;
if (gid == 0) {
  for (j=0; j < n_cols; ++j){
    printf("### %f\n", sorted_coords[j*n_rows+i_ref]);
  }
}
    if (gid+offset < n_rows) {
      // compute squared dist
      for (j=0; j < n_cols; ++j) {
        c = coords[j*n_rows+i_ref] - sorted_coords[j*n_rows + gid + offset];
        dist2 = fma(c, c, dist2);
      }
      // write results: 1.0 if in radius, 0.0 if not
      for (r=0; r < n_radii; ++r) {
//TODO
        if (dist2 <= radii2[j]) {
          in_radius[r*n_rows + gid] = dist2;
        } else {
          in_radius[r*n_rows + gid] = dist2;
        }
      }
    }
  }

  Pops
  calculate_populations_partial(const float* coords
                              , const std::vector<float>& sorted_coords
                              , const std::vector<float>& blimits
                              , std::size_t n_rows
                              , std::size_t n_cols
                              , std::vector<float> radii
                              , std::size_t i_from
                              , std::size_t i_to
                              , int i_gpu) {
    ASSUME_ALIGNED(coords);

    for (int ii=0; ii < n_cols; ++ii) {
      std::cerr << "@ " << coords[ii*n_rows] << std::endl;
      std::cerr << "@s " << sorted_coords[ii*n_rows] << std::endl;
    }


    unsigned int n_rows_ext = Tools::min_multiplicator(n_rows, BSIZE) * BSIZE;
    unsigned int n_radii = radii.size();
    // make sure radii are in descending order
    std::sort(radii.begin(), radii.end(), std::greater<float>());
    // setup device & streams
    hipSetDevice(i_gpu);
    hipStream_t streams[N_STREAMS];
    for (unsigned int s=0; s < N_STREAMS; ++s) {
      hipStreamCreate(&streams[s]);
    }
    // copy coords to device
    float* d_coords;
    hipMalloc((void**) &d_coords
             , sizeof(float) * n_rows * n_cols);
    hipMemcpy(d_coords
             , coords
             , sizeof(float) * n_rows * n_cols
             , hipMemcpyHostToDevice);
    float* d_sorted_coords;
    hipMalloc((void**) &d_sorted_coords
             , sizeof(float) * n_rows * n_cols);
    hipMemcpy(d_sorted_coords
             , sorted_coords.data()
             , sizeof(float) * n_rows * n_cols
             , hipMemcpyHostToDevice);
    // copy squared radii to device
    float* d_radii2;
    hipMalloc((void**) &d_radii2
             , sizeof(float) * n_radii);
    std::vector<float> radii2(radii);
    for (float& r: radii2) {
      r *= r;
    }
    hipMemcpy(d_radii2
             , radii2.data()
             , sizeof(float) * n_radii
             , hipMemcpyHostToDevice);
    // tmp buffer for in/out info & reference coords (per stream)
    float* d_in_radius[N_STREAMS];
    for (unsigned int s=0; s < N_STREAMS; ++s) {
      hipMalloc((void**) &d_in_radius[s]
               , sizeof(float) * n_rows * n_radii);
    }
    // result buffer
    float* d_pops;
    hipMalloc((void**) &d_pops
             , sizeof(float) * n_rows * n_radii);
    hipMemset(d_pops
             , 0
             , sizeof(float) * n_rows * n_radii);
    // populations per frame
    for (std::size_t i=i_from; i < i_to; ++i) {
      unsigned int i_stream = i % N_STREAMS;
      //  initialize d_in_radius to 0 -> per default not in hypersphere
      hipMemset(d_in_radius[i_stream]
               , 0
               , sizeof(float) * n_rows * n_radii);
      // prune range for faster computation
      // (using largest radius in first dimension)
//      auto min_max_box = Clustering::Tools::min_max_box(blimits
//                                                      , coords[i]
//                                                      , radii[0]);
//
//      unsigned int offset = min_max_box.first * BSIZE;
//      unsigned int rng = (min_max_box.second-min_max_box.first+1);


      //TODO
      unsigned int offset = 0;
      unsigned int rng = n_rows_ext/BSIZE;

      in_radius <<< rng
                  , BSIZE
                  , 0
                  , streams[i_stream] >>> (offset
                                         , d_sorted_coords
                                         , d_coords
                                         , i
                                         , n_rows
                                         , n_cols
                                         , d_radii2
                                         , n_radii
                                         , d_in_radius[i_stream]);
      //TODO
      check_error();

      hipDeviceSynchronize();
      if (i == 0) {
        std::vector<float> tmp_in_rad(n_radii*n_rows);
        hipMemcpy(tmp_in_rad.data()
                 , d_in_radius[i_stream]
                 , sizeof(float) * n_rows * n_radii
                 , hipMemcpyDeviceToHost);
        for (auto f: tmp_in_rad) {
          std::cout << f << std::endl;
        }
        exit(EXIT_FAILURE);
      }





      // compute pops per radius
      for (unsigned int r=0; r < n_radii; ++r) {
        // pops stored col-wise -> just set an offset ...
        offset = r*n_rows;
        //TODO stupid: don't run over all rows for reduction, use boxlimits!
        reduce_sum<BSIZE> <<< n_rows_ext/BSIZE
                            , BSIZE
                            , 0
                            , streams[i_stream] >>> (offset
                                                   , d_in_radius[i_stream]
                                                   , n_rows
                                                   , d_pops
                                                   , r*n_rows + i);
        //TODO
        check_error();
      }
    }
    hipDeviceSynchronize();
    // retrieve pops
    std::vector<float> tmp_pops(n_rows*n_radii);
    hipMemcpy(tmp_pops.data()
             , d_pops
             , sizeof(float) * n_rows * n_radii
             , hipMemcpyDeviceToHost);
    // sort tmp_pops into pops
    Pops pops;
    for (unsigned int r=0; r < n_radii; ++r) {
      pops[radii[r]].resize(n_rows, 0);
      for (unsigned int i=i_from; i < i_to; ++i) {
        pops[radii[r]][i] = tmp_pops[r*n_rows+i];
      }
    }
    hipFree(d_sorted_coords);
    hipFree(d_radii2);
    hipFree(d_pops);
    for (unsigned int s=0; s < N_STREAMS; ++s) {
      hipFree(d_in_radius[s]);
    }
    return pops;
  }

  Pops
  calculate_populations(const float* coords
                      , const std::size_t n_rows
                      , const std::size_t n_cols
                      , std::vector<float> radii) {
//    ASSUME_ALIGNED(coords);
    for (int ii=0; ii < n_cols; ++ii) {
      std::cerr << "@@ " << coords[ii*n_rows] << std::endl;
    }




    std::sort(radii.begin(), radii.end(), std::greater<float>());
    std::size_t n_radii = radii.size();
    std::vector<float> rad2(n_radii);
    for (std::size_t i=0; i < n_radii; ++i) {
      rad2[i] = radii[i]*radii[i];
    }
    // sort coordinates on first dimension for neighbor pruning
    std::vector<float> sorted_coords = Clustering::Tools::dim1_sorted_coords(coords
                                                                           , n_rows
                                                                           , n_cols);
    // box limits for pruning
    std::vector<float> blimits = Clustering::Tools::boxlimits(sorted_coords
                                                            , BSIZE
                                                            , n_cols);
    int n_gpus;
    hipGetDeviceCount(&n_gpus);
    if (n_gpus == 0) {
      std::cerr << "error: no CUDA-compatible GPUs found" << std::endl;
      exit(EXIT_FAILURE);
    }

//TODO
    n_gpus = 1;


    int gpu_range = n_rows / n_gpus;
    int i;
    std::vector<Pops> partial_pops(n_gpus);
    #pragma omp parallel for default(none)\
                             private(i)\
                             firstprivate(n_gpus,n_rows,n_cols,gpu_range)\
                             shared(partial_pops,radii,coords,sorted_coords,blimits)\
                             num_threads(1)
    for (i=0; i < n_gpus; ++i) {
      // compute partial populations in parallel
      // on all available GPUs
      partial_pops[i] = calculate_populations_partial(coords
                                                    , sorted_coords
                                                    , blimits
                                                    , n_rows
                                                    , n_cols
                                                    , radii
                                                    , i*gpu_range
                                                    , i == (n_gpus-1)
                                                        ? n_rows
                                                        : (i+1)*gpu_range
                                                    , i);
    }
    Pops pops;
    // combine pops
    for (float r: radii) {
      pops[r].resize(n_rows, 0);
      for (i=0; i < n_rows; ++i) {
        for (unsigned int i_gpu=0; i_gpu < n_gpus; ++i_gpu) {
          pops[r][i] += partial_pops[i_gpu][r][i];
        }
      }
    }
    return pops;
  }

}}} // end Clustering::Density::CUDA

