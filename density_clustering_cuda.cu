#include "hip/hip_runtime.h"

#include "tools.hpp"
#include "density_clustering_cuda.hpp"
#include "logger.hpp"

#include <algorithm>

#include <hip/hip_runtime.h>
#include <omp.h>

#include "lts_cuda_kernels.cuh"

// for pops
#define BSIZE_POPS 512
//#define BSIZE_POPS 1024

// for neighborhood search
#define BSIZE_NH 128
#define N_STREAMS_NH 1

// for screening
#define BSIZE_SCR 512

namespace Clustering {
namespace Density {
namespace CUDA {

  __global__ void
  population_count(unsigned int offset
                 , float* coords
                 , unsigned int n_rows
                 , unsigned int n_cols
                 , float* radii2
                 , unsigned int n_radii
                 , unsigned int* pops
                 , unsigned int i_from
                 , unsigned int i_to) {
    extern __shared__ float smem[];
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int bsize = blockDim.x;
    unsigned int gid = bid * bsize + tid + i_from;
    // load frames for comparison into shared memory
    int comp_size = min(bsize, n_rows - offset);
    if (tid < comp_size) {
      for (unsigned int j=0; j < n_cols; ++j) {
        smem[tid*n_cols+j] = coords[(tid+offset)*n_cols+j];
      }
    }
    __syncthreads();
    // count neighbors
    if (gid < i_to) {
      unsigned int ref_id = tid+bsize;
      // load reference coordinates for re-use into shared memory
      for (unsigned int j=0; j < n_cols; ++j) {
        smem[ref_id*n_cols+j] = coords[gid*n_cols+j];
      }
      for (unsigned int r=0; r < n_radii; ++r) {
        unsigned int local_pop = 0;
        float rad2 = radii2[r];
        for (unsigned int i=0; i < comp_size; ++i) {
          float dist2 = 0.0f;
          for (unsigned int j=0; j < n_cols; ++j) {
            float c = smem[ref_id*n_cols+j] - smem[i*n_cols+j];
            dist2 = fma(c, c, dist2);
          }
          if (dist2 <= rad2) {
            ++local_pop;
          }
        }
        // update frame populations (per radius)
        pops[r*n_rows+gid] += local_pop;
      }
    }
  }

  __global__ void
  nearest_neighbor_search(unsigned int offset
                        , float* coords
                        , unsigned int n_rows
                        , unsigned int n_cols
                        , float* fe
                        , float* nh_dist_ndx
                        , float* nhhd_dist_ndx
                        , unsigned int i_from
                        , unsigned int i_to) {
    extern __shared__ float smem[];
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int bsize = blockDim.x;
    unsigned int gid = bid * bsize + tid + i_from;

    float nh_mindist;
    float nh_minndx;
    float nhhd_mindist;
    float nhhd_minndx;
    float ref_fe;
    unsigned int ref_id;

    // load frames for comparison into shared memory
    int comp_size = min(bsize, n_rows - offset);
    if (tid < comp_size) {
      for (unsigned int j=0; j < n_cols; ++j) {
        smem[tid*n_cols+j] = coords[(tid+offset)*n_cols+j];
      }
    }
    __syncthreads();

    if (gid < i_to) {
      ref_id = tid+bsize;
      // load reference coordinates for re-use into shared memory
      for (unsigned int j=0; j < n_cols; ++j) {
        smem[ref_id*n_cols+j] = coords[gid*n_cols+j];
      }
      ref_fe = fe[gid];
      // load current best mindists into registers
      nh_mindist = nh_dist_ndx[gid];
      nh_minndx = nh_dist_ndx[n_rows+gid];
      nhhd_mindist = nhhd_dist_ndx[gid];
      nhhd_minndx = nhhd_dist_ndx[n_rows+gid];
      // compare squared distances of reference
      // compare squared distances of reference
      // to (other) frames in shared mem
      for (unsigned int i=0; i < comp_size; ++i) {
        float dist2=0.0f;
        for (unsigned int j=0; j < n_cols; ++j) {
          float c = smem[ref_id*n_cols+j] - smem[i*n_cols+j];
          dist2 = fma(c, c, dist2);
        }
        // frame with min distance (i.e. nearest neighbor)
        if ((nh_mindist == 0)
         || (dist2 < nh_mindist && dist2 != 0)) {
          nh_mindist = dist2;
          nh_minndx = i+offset;
        }
        // frame with min distance and lower energy
        if ((nhhd_mindist == 0 && fe[i+offset] < ref_fe)
         || (dist2 < nhhd_mindist && fe[i+offset] < ref_fe && dist2 != 0)) {
          nhhd_mindist = dist2;
          nhhd_minndx = i+offset;
        }
      }
      // write results (dist & ndx) to global buffers
      nh_dist_ndx[gid] = nh_mindist;
      nh_dist_ndx[n_rows+gid] = nh_minndx;
      nhhd_dist_ndx[gid] = nhhd_mindist;
      nhhd_dist_ndx[n_rows+gid] = nhhd_minndx;
    }
  }

  ////

  void
  check_error(std::string msg) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "CUDA error: "
                << msg << "\n"
                << hipGetErrorString(err) << std::endl;
      exit(EXIT_FAILURE);
    }
  }

  int
  get_num_gpus() {
    int n_gpus;
    hipGetDeviceCount(&n_gpus);
    check_error("trying to get number of available GPUs");
    if (n_gpus == 0) {
      std::cerr << "error: no CUDA-compatible GPUs found" << std::endl;
      exit(EXIT_FAILURE);
    } else {
      return n_gpus;
    }
  }

  Pops
  calculate_populations_per_gpu(const float* coords
                              , std::size_t n_rows
                              , std::size_t n_cols
                              , std::vector<float> radii
                              , std::size_t i_from
                              , std::size_t i_to
                              , int i_gpu) {
    using Clustering::Tools::min_multiplicator;
    ASSUME_ALIGNED(coords);
    unsigned int n_radii = radii.size();
    std::vector<float> rad2(n_radii);
    for (std::size_t i=0; i < n_radii; ++i) {
      rad2[i] = radii[i]*radii[i];
    }
    // GPU setup
    hipSetDevice(i_gpu);
    float* d_coords;
    float* d_rad2;
    unsigned int* d_pops;
    hipMalloc((void**) &d_coords
             , sizeof(float) * n_rows * n_cols);
    hipMalloc((void**) &d_pops
             , sizeof(unsigned int) * n_rows * n_radii);
    hipMalloc((void**) &d_rad2
             , sizeof(float) * n_radii);
    check_error("pop-calc device mallocs");
    hipMemset(d_pops
             , 0
             , sizeof(unsigned int) * n_rows * n_radii);
    check_error("pop-calc memset");
    hipMemcpy(d_coords
             , coords
             , sizeof(float) * n_rows * n_cols
             , hipMemcpyHostToDevice);
    hipMemcpy(d_rad2
             , rad2.data()
             , sizeof(float) * n_radii
             , hipMemcpyHostToDevice);
    check_error("pop-calc mem copies");
    int max_shared_mem;
    hipDeviceGetAttribute(&max_shared_mem
                         , hipDeviceAttributeMaxSharedMemoryPerBlock
                         , i_gpu);
    check_error("getting max shared mem size");
    unsigned int block_size = BSIZE_POPS;
    unsigned int shared_mem = 2 * block_size * n_cols * sizeof(float);
    if (shared_mem > max_shared_mem) {
      std::cerr << "error: max. shared mem per block too small on this GPU.\n"
                << "       either reduce BSIZE_POPS or get a better GPU."
                << std::endl;
      exit(EXIT_FAILURE);
    }
    unsigned int block_rng = min_multiplicator(i_to-i_from, block_size);
    Clustering::logger(std::cout) << "# blocks needed: "
                                  << block_rng << std::endl;
    for (unsigned int i=0; i*block_size < n_rows; ++i) {
      population_count <<< block_rng
                         , block_size
                         , shared_mem >>> (i*block_size
                                         , d_coords
                                         , n_rows
                                         , n_cols
                                         , d_rad2
                                         , n_radii
                                         , d_pops
                                         , i_from
                                         , i_to);
    }
    hipDeviceSynchronize();
    check_error("after kernel loop");
    // get partial results from GPU
    std::vector<unsigned int> partial_pops(n_rows*n_radii);
    hipMemcpy(partial_pops.data()
             , d_pops
             , sizeof(unsigned int) * n_rows * n_radii
             , hipMemcpyDeviceToHost);
    // sort into resulting pops
    Pops pops;
    for (unsigned int r=0; r < n_radii; ++r) {
      pops[radii[r]].resize(n_rows, 0);
      for (unsigned int i=i_from; i < i_to; ++i) {
        pops[radii[r]][i] = partial_pops[r*n_rows+i];
      }
    }
    hipFree(d_coords);
    hipFree(d_rad2);
    hipFree(d_pops);
    return pops;
  }

  Pops
  calculate_populations(const float* coords
                      , const std::size_t n_rows
                      , const std::size_t n_cols
                      , std::vector<float> radii) {
    using Clustering::Tools::dim1_sorted_coords;
    using Clustering::Tools::boxlimits;
    ASSUME_ALIGNED(coords);
    std::sort(radii.begin(), radii.end(), std::greater<float>());
    int n_gpus = get_num_gpus();
    int gpu_range = n_rows / n_gpus;
    int i;
    std::vector<Pops> partial_pops(n_gpus);
    #pragma omp parallel for default(none)\
      private(i)\
      firstprivate(n_gpus,n_rows,n_cols,gpu_range)\
      shared(partial_pops,radii,coords)\
      num_threads(n_gpus)\
      schedule(dynamic,1)
    for (i=0; i < n_gpus; ++i) {
      // compute partial populations in parallel
      // on all available GPUs
      partial_pops[i] = calculate_populations_per_gpu(coords
                                                    , n_rows
                                                    , n_cols
                                                    , radii
                                                    , i*gpu_range
                                                    , i == (n_gpus-1)
                                                        ? n_rows
                                                        : (i+1)*gpu_range
                                                    , i);
    }
    Pops pops;
    // combine pops
    for (float r: radii) {
      pops[r].resize(n_rows, 0);
      for (i=0; i < n_rows; ++i) {
        for (unsigned int i_gpu=0; i_gpu < n_gpus; ++i_gpu) {
          pops[r][i] += partial_pops[i_gpu][r][i];
        }
      }
    }
    return pops;
  }

  std::tuple<Neighborhood, Neighborhood>
  nearest_neighbors_per_gpu(const float* coords
                          , const std::size_t n_rows
                          , const std::size_t n_cols
                          , const std::vector<float>& free_energy
                          , std::size_t i_from
                          , std::size_t i_to
                          , int i_gpu) {
    using Clustering::Tools::min_multiplicator;
    ASSUME_ALIGNED(coords);
    // GPU setup
    hipSetDevice(i_gpu);
    hipStream_t streams[N_STREAMS_NH];
    float* d_coords;
    float* d_fe;
    float* d_nh[N_STREAMS_NH];
    float* d_nhhd[N_STREAMS_NH];
    hipMalloc((void**) &d_coords
             , sizeof(float) * n_rows * n_cols);
    hipMalloc((void**) &d_fe
             , sizeof(float) * n_rows);
    for (unsigned int i=0; i < N_STREAMS_NH; ++i) {
      hipMalloc((void**) &d_nh[i]
               , sizeof(float) * n_rows * 2);
      hipMalloc((void**) &d_nhhd[i]
               , sizeof(float) * n_rows * 2);
      hipMemset(d_nh[i]
               , 0
               , sizeof(float) * n_rows * 2);
      hipMemset(d_nhhd[i]
               , 0
               , sizeof(float) * n_rows * 2);
      hipStreamCreate(&streams[i]);
    }
    hipMemcpy(d_coords
             , coords
             , sizeof(float) * n_rows * n_cols
             , hipMemcpyHostToDevice);
    hipMemcpy(d_fe
             , free_energy.data()
             , sizeof(float) * n_rows
             , hipMemcpyHostToDevice);
    int max_shared_mem;
    hipDeviceGetAttribute(&max_shared_mem
                         , hipDeviceAttributeMaxSharedMemoryPerBlock
                         , i_gpu);
    check_error("retrieving max shared mem");
    unsigned int block_size = BSIZE_NH;
    unsigned int shared_mem = 2 * block_size * n_cols * sizeof(float);
    if (shared_mem > max_shared_mem) {
      std::cerr << "error: max. shared mem per block too small on this GPU.\n"
                << "       either reduce block_size for NN search or get a "
                <<        "better GPU." << std::endl;
      exit(EXIT_FAILURE);
    }
    unsigned int block_rng = min_multiplicator(i_to-i_from, block_size);
    for (unsigned int i=0; i*block_size < n_rows; ++i) {
      unsigned int i_stream = i % N_STREAMS_NH;
      nearest_neighbor_search <<< block_rng
                                , block_size
                                , shared_mem
                                , streams[i_stream] >>> (i*block_size
                                                       , d_coords
                                                       , n_rows
                                                       , n_cols
                                                       , d_fe
                                                       , d_nh[i_stream]
                                                       , d_nhhd[i_stream]
                                                       , i_from
                                                       , i_to);
    }
    hipDeviceSynchronize();
    check_error("after kernel loop");
    // initialize neighborhoods
    Neighborhood nh;
    Neighborhood nhhd;
    for (unsigned int i=0; i < n_rows; ++i) {
      nh[i] = {i, std::numeric_limits<float>::max()};
      nhhd[i] = {i, std::numeric_limits<float>::max()};
    }
    // collect partial results from streams
    for (unsigned int i_stream=0; i_stream < N_STREAMS_NH; ++i_stream) {
      std::vector<float> dist_ndx(n_rows * 2);
      auto update_nh = [&dist_ndx,n_rows] (Neighborhood& _nh) -> void {
        for (unsigned int i=0; i < n_rows; ++i) {
          if (dist_ndx[i] < _nh[i].second && dist_ndx[i] != 0) {
            _nh[i] = {(unsigned int) dist_ndx[n_rows+i]
                    , dist_ndx[i]};
          }
        }
      };
      hipMemcpy(dist_ndx.data()
               , d_nh[i_stream]
               , sizeof(float) * n_rows * 2
               , hipMemcpyDeviceToHost);
      update_nh(nh);
      hipMemcpy(dist_ndx.data()
               , d_nhhd[i_stream]
               , sizeof(float) * n_rows * 2
               , hipMemcpyDeviceToHost);
      update_nh(nhhd);
    }
    // device cleanup
    hipFree(d_coords);
    hipFree(d_fe);
    for (unsigned int i=0; i < N_STREAMS_NH; ++i) {
      hipFree(d_nh[i]);
      hipFree(d_nhhd[i]);
    }
    return std::make_tuple(nh, nhhd);
  }

  std::tuple<Neighborhood, Neighborhood>
  nearest_neighbors(const float* coords
                  , const std::size_t n_rows
                  , const std::size_t n_cols
                  , const std::vector<float>& free_energy) {
    int n_gpus = get_num_gpus();
    std::vector<std::tuple<Neighborhood, Neighborhood>> partials(n_gpus);
    unsigned int gpu_range = n_rows / n_gpus;
    unsigned int i_gpu;
    #pragma omp parallel for default(none)\
      private(i_gpu)\
      firstprivate(n_gpus,n_rows,n_cols,gpu_range)\
      shared(partials,coords,free_energy)\
      num_threads(n_gpus)
    for (i_gpu=0; i_gpu < n_gpus; ++i_gpu) {
      partials[i_gpu] = nearest_neighbors_per_gpu(coords
                                                , n_rows
                                                , n_cols
                                                , free_energy
                                                , i_gpu*gpu_range
                                                , i_gpu == (n_gpus-1)
                                                        ? n_rows
                                                        : (i_gpu+1)*gpu_range
                                                , i_gpu);
    }
    // combine partial neighborhood results from different gpus
    Neighborhood nh;
    Neighborhood nhhd;
    std::tie(nh, nhhd) = partials[0];
    for (i_gpu=1; i_gpu < n_gpus; ++i_gpu) {
      Neighborhood partial_nh;
      Neighborhood partial_nhhd;
      std::tie(partial_nh, partial_nhhd) = partials[i_gpu];
      for (unsigned int i=0; i < n_rows; ++i) {
        if (partial_nh[i].second < nh[i].second) {
          nh[i] = partial_nh[i];
        }
        if (partial_nhhd[i].second < nhhd[i].second) {
          nhhd[i] = partial_nhhd[i];
        }
      }
    }
    return std::make_tuple(nh, nhhd);
  }


  __global__ void
  initial_density_clustering_krnl(unsigned int offset
                                , float* sorted_coords
                                , unsigned int n_rows
                                , unsigned int n_cols
                                , float max_dist2
                                , unsigned int* clustering
                                , unsigned int i_from
                                , unsigned int i_to) {
    extern __shared__ float smem[];
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int bsize = blockDim.x;
    unsigned int gid = bid * bsize + tid + i_from;
    // load frames for comparison into shared memory
    int comp_size = min(bsize, n_rows - offset);
    if (tid < comp_size) {
      for (unsigned int j=0; j < n_cols; ++j) {
        smem[tid*n_cols+j] = sorted_coords[(tid+offset)*n_cols+j];
      }
    }
    __syncthreads();
    if (gid < i_to) {
      unsigned int self_id = gid + 1;
      unsigned int cluster_id = clustering[gid];
      if (cluster_id == 0) {
        cluster_id = self_id;
      }
      unsigned int ref_id = tid+bsize;
      // load reference coordinates for re-use into shared memory
      for (unsigned int j=0; j < n_cols; ++j) {
        smem[ref_id*n_cols+j] = sorted_coords[gid*n_cols+j];
      }
      for (unsigned int i=0; i < comp_size; ++i) {
        float dist2 = 0.0f;
        for (unsigned int j=0; j < n_cols; ++j) {
          float c = smem[ref_id*n_cols+j] - smem[i*n_cols+j];
          dist2 = fma(c, c, dist2);
        }
        if (dist2 < max_dist2) {
          unsigned int min_id = min(cluster_id, i+offset+1);
          if (min_id == self_id) {
            cluster_id = max(cluster_id, i+offset+1);
          } else {
            cluster_id = min_id;
          }
        }
      }
      clustering[gid] = cluster_id;
    }
  }

//  unsigned int
//  follow(const std::vector<unsigned int>& clustering
//       , unsigned int id
//       , unsigned int min_id = 0) {
//    if (id == 0
//     || id == min_id) {
//      return id;
//    } else {
//      return follow(clustering
//                  , clustering[id-1]
//                  , (min_id == 0)
//                      ? id
//                      : std::min(id, min_id));
//    }
//  }

  std::vector<unsigned int>
  sanitize_state_names(std::vector<unsigned int> clustering) {

    //TODO: lump frames into microstates.
    //      be careful: cluster ids do not necessarily
    //                  denote frame ids (especially not with previous
    //                  clustering)

    // map names to ascending index
    unsigned int id=0;
    std::map<unsigned int, unsigned int> name_dict;
    for (unsigned int name: unique_names) {
      ++id;
      name_dict[name] = id;
    }
    // relabel states
    for (unsigned int& i: clustering) {
      i = name_dict[i];
    }
    return clustering;
  }

//TODO remove old code
//    for (unsigned int i=0; i < clustering.size(); ++i) {
//      clustering[i] = follow(clustering
//                           , clustering[i]);
//    }
//    return clustering;
//  }


  std::vector<std::size_t>
  initial_density_clustering(const std::vector<float>& free_energy
                           , const Neighborhood& nh
                           , const float free_energy_threshold
                           , const float* coords
                           , const std::size_t n_rows
                           , const std::size_t n_cols
                           , const std::vector<std::size_t> initial_clusters) {
    using Clustering::Tools::min_multiplicator;
    std::size_t first_frame_above_threshold;
    double sigma2;
    std::vector<FreeEnergy> fe_sorted;
    std::vector<std::size_t> prev_clustering;
    std::size_t prev_max_state;
    // data preparation
    std::tie(prev_clustering
           , first_frame_above_threshold
           , sigma2
           , fe_sorted
           , std::ignore
           , prev_max_state) = prepare_initial_clustering(free_energy
                                                        , nh
                                                        , free_energy_threshold
                                                        , n_rows
                                                        , initial_clusters);
    // write log
//    screening_log(sigma2
//                , first_frame_above_threshold
//                , fe_sorted);
    float max_dist2 = 4*sigma2;
    // prepare CUDA environment
    int n_gpus = get_num_gpus();
    std::vector<float*> d_coords_sorted(n_gpus);
    std::vector<unsigned int*> d_clustering(n_gpus);
    // sort coords (and previous clustering results)
    // according to free energies
    std::vector<float> tmp_coords_sorted(n_rows * n_cols);
    std::vector<unsigned int> prev_clustering_sorted(n_rows);
    for (unsigned int i=0; i < n_rows; ++i) {
      for (unsigned int j=0; j < n_cols; ++j) {
        tmp_coords_sorted[i*n_cols+j] = coords[fe_sorted[i].first*n_cols+j];
      }
      prev_clustering_sorted[i] = prev_clustering[fe_sorted[i].first];
    }
    unsigned int block_rng;
//TODO: check: what happens if no new frames are added compared to prev run?
    unsigned int i_from;
    unsigned int i_to;
    unsigned int i;
    // re-use initial clustering results
    unsigned int prev_last_frame = 0;
    if (prev_clustering_sorted[0] != 0) {
      for (i=1; i < n_rows; ++i) {
        if (prev_clustering_sorted[i] == 0) {
          prev_last_frame = i;
          break;
        }
      }
    }
    int i_gpu;
    unsigned int gpu_rng =
      min_multiplicator(first_frame_above_threshold - prev_last_frame
                      , n_gpus);
    if (gpu_rng == 0) {
      // nothing to do, since all frames below threshold were already
      // below previous threshold
      return initial_clusters;
    }
    int max_shared_mem;
    // assuming GPUs are of same type with same amount of memory
    hipDeviceGetAttribute(&max_shared_mem
                         , hipDeviceAttributeMaxSharedMemoryPerBlock
                         , 0);
    check_error("getting max shared mem size");
    unsigned int shared_mem = 2 * BSIZE_SCR * n_cols * sizeof(float);
    #pragma omp parallel for\
      default(none)\
      private(i,i_gpu,block_rng,i_from,i_to)\
      firstprivate(n_gpus,n_rows,n_cols,gpu_rng,max_dist2,\
                   prev_last_frame,prev_max_state,\
                   shared_mem,first_frame_above_threshold)\
      shared(d_coords_sorted,d_clustering,\
             tmp_coords_sorted,prev_clustering_sorted)\
      num_threads(n_gpus)
    for (i_gpu=0; i_gpu < n_gpus; ++i_gpu) {
      // allocate memory and copy sorted coords to GPUs
      hipSetDevice(i_gpu);
      hipMalloc((void**) &d_coords_sorted[i_gpu]
               , sizeof(float) * n_rows * n_cols);
      hipMalloc((void**) &d_clustering[i_gpu]
               , sizeof(unsigned int) * n_rows);
      hipMemcpy(d_coords_sorted[i_gpu]
               , tmp_coords_sorted.data()
               , sizeof(float) * n_rows * n_cols
               , hipMemcpyHostToDevice);


      //TODO change kernel to use two lists: own state + min next-neighbor state


      // copy prev results to GPU-buffer (and set the rest to zero)
      hipMemset(d_clustering[i_gpu]
               , 0
               , sizeof(unsigned int) * n_rows);
      hipMemcpy(d_clustering[i_gpu]
               , prev_clustering_sorted.data()
               , sizeof(unsigned int) * n_rows
               , hipMemcpyHostToDevice);
      // perform initial clustering on yet unclustered frames
      i_from = prev_last_frame + i_gpu * gpu_rng;
      i_to = (i_gpu == (n_gpus-1))
           ? first_frame_above_threshold
           : prev_last_frame + (i_gpu+1) * gpu_rng;
      block_rng = min_multiplicator(i_to-i_from
                                  , BSIZE_SCR);
      for (i=0; i*BSIZE_SCR < first_frame_above_threshold; ++i) {
        initial_density_clustering_krnl
          <<< block_rng
            , BSIZE_SCR
            , shared_mem >>>
          (i*BSIZE_SCR
         , d_coords_sorted[i_gpu]
         , n_rows
         , n_cols
         , max_dist2
         , d_clustering[i_gpu]
         , i_from
         , i_to);
      }
      hipDeviceSynchronize();
      check_error("after kernel loop");
    }
    // collect & merge clustering results from GPUs
    std::vector<unsigned int> clustering_sorted = prev_clustering_sorted;
    for (int i_gpu=0; i_gpu < n_gpus; ++i_gpu) {
      std::vector<unsigned int> tmp_clust(n_rows, 0);
      hipMemcpy(tmp_clust.data()
               , d_clustering[i_gpu]
               , sizeof(unsigned int) * first_frame_above_threshold
               , hipMemcpyDeviceToHost);
      for (i=prev_last_frame; i < first_frame_above_threshold; ++i) {
        if (i_gpu == 0) {
          clustering_sorted[i] = tmp_clust[i];
        } else {
          clustering_sorted[i] = std::max(clustering_sorted[i]
                                        , tmp_clust[i]);
        }
      }
    }
    // reduce clustering to min number of ids
    clustering_sorted = sanitize_state_names(clustering_sorted);
    //TODO: refined clustering (by rerunning with alternative kernel)




    // convert state trajectory from
    // FE-sorted order to original order
    std::vector<std::size_t> clustering(n_rows, 0);
    for (unsigned int i=0; i < n_rows; ++i) {
      clustering[fe_sorted[i].first] = clustering_sorted[i];
    }
    // cleanup CUDA environment
    for (int i_gpu=0; i_gpu < n_gpus; ++i_gpu) {
      hipFree(d_coords_sorted[i_gpu]);
      hipFree(d_clustering[i_gpu]);
    }
    return normalized_cluster_names(first_frame_above_threshold
                                  , clustering
                                  , fe_sorted);
  }

}}} // end Clustering::Density::CUDA

