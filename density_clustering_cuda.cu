#include "hip/hip_runtime.h"

#include "tools.hpp"
#include "density_clustering_cuda.hpp"
#include "logger.hpp"

#include <algorithm>

#include <hip/hip_runtime.h>
#include <omp.h>

#include "lts_cuda_kernels.cuh"

// for pops
//#define BSIZE_POPS 128
#define BSIZE_POPS 1024

// for neighborhood search
#define BSIZE_NH 128
#define N_STREAMS_NH 1

namespace Clustering {
namespace Density {
namespace CUDA {

  __global__ void
  population_count(unsigned int offset
                 , float* coords
                 , unsigned int n_rows
                 , unsigned int n_cols
                 , float* radii2
                 , unsigned int n_radii
                 , unsigned int* pops
                 , unsigned int i_from
                 , unsigned int i_to) {
    extern __shared__ float smem[];
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int bsize = blockDim.x;
    unsigned int gid = bid * bsize + tid + i_from;
    // load frames for comparison into shared memory
    int comp_size = min(bsize, n_rows - offset);
    if (tid < comp_size) {
      for (unsigned int j=0; j < n_cols; ++j) {
        smem[tid*n_cols+j] = coords[(tid+offset)*n_cols+j];
      }
    }
    __syncthreads();
    // count neighbors
    if (gid < i_to) {
      unsigned int ref_id = tid+bsize;
      // load reference coordinates for re-use into shared memory
      for (unsigned int j=0; j < n_cols; ++j) {
        smem[ref_id*n_cols+j] = coords[gid*n_cols+j];
      }
      for (unsigned int r=0; r < n_radii; ++r) {
        unsigned int local_pop = 0;
        float rad2 = radii2[r];
        for (unsigned int i=0; i < comp_size; ++i) {
          float dist2 = 0.0f;
          for (unsigned int j=0; j < n_cols; ++j) {
            float c = smem[ref_id*n_cols+j] - smem[i*n_cols+j];
            dist2 = fma(c, c, dist2);
          }
          if (dist2 <= rad2) {
            ++local_pop;
          }
        }
        // update frame populations (per radius)
        pops[r*n_rows+gid] += local_pop;
      }
    }
  }

  __global__ void
  nearest_neighbor_search(unsigned int offset
                        , float* coords
                        , unsigned int n_rows
                        , unsigned int n_cols
                        , float* fe
                        , float* nh_dist_ndx
                        , float* nhhd_dist_ndx
                        , unsigned int i_from
                        , unsigned int i_to) {
    extern __shared__ float smem[];
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int bsize = blockDim.x;
    unsigned int gid = bid * bsize + tid + i_from;

    float nh_mindist;
    float nh_minndx;
    float nhhd_mindist;
    float nhhd_minndx;
    float ref_fe;
    unsigned int ref_id;

    // load frames for comparison into shared memory
    int comp_size = min(bsize, n_rows - offset);
    if (tid < comp_size) {
      for (unsigned int j=0; j < n_cols; ++j) {
        smem[tid*n_cols+j] = coords[(tid+offset)*n_cols+j];
      }
    }
    __syncthreads();

    if (gid < i_to) {
      ref_id = tid+bsize;
      // load reference coordinates for re-use into shared memory
      for (unsigned int j=0; j < n_cols; ++j) {
        smem[ref_id*n_cols+j] = coords[gid*n_cols+j];
      }
      ref_fe = fe[gid];
      // load current best mindists into registers
      nh_mindist = nh_dist_ndx[gid];
      nh_minndx = nh_dist_ndx[n_rows+gid];
      nhhd_mindist = nhhd_dist_ndx[gid];
      nhhd_minndx = nhhd_dist_ndx[n_rows+gid];
      // compare squared distances of reference
      // to (other) frames in shared mem
      for (unsigned int i=0; i < comp_size; ++i) {
        float dist2=0.0f;
        for (unsigned int j=0; j < n_cols; ++j) {
          float c = smem[ref_id*n_cols+j] - smem[i*n_cols+j];
          dist2 = fma(c, c, dist2);
        }
        // frame with min distance (i.e. nearest neighbor)
        if ((nh_mindist == 0)
         || (dist2 < nh_mindist && dist2 != 0)) {
          nh_mindist = dist2;
          nh_minndx = i+offset;
        }
        // frame with min distance and lower energy
        if ((nhhd_mindist == 0 && fe[i+offset] < ref_fe)
         || (dist2 < nhhd_mindist && fe[i+offset] < ref_fe && dist2 != 0)) {
          nhhd_mindist = dist2;
          nhhd_minndx = i+offset;
        }
      }
      // write results (dist & ndx) to global buffers
      nh_dist_ndx[gid] = nh_mindist;
      nh_dist_ndx[n_rows+gid] = nh_minndx;
      nhhd_dist_ndx[gid] = nhhd_mindist;
      nhhd_dist_ndx[n_rows+gid] = nhhd_minndx;
    }
  }

  ////

  void check_error(std::string msg="") {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "CUDA error: "
                << msg << "\n"
                << hipGetErrorString(err) << std::endl;
      exit(EXIT_FAILURE);
    }
  }

  int
  get_num_gpus() {
    int n_gpus;
    hipGetDeviceCount(&n_gpus);
    if (n_gpus == 0) {
      std::cerr << "error: no CUDA-compatible GPUs found" << std::endl;
      exit(EXIT_FAILURE);
    } else {
      return n_gpus;
    }
  }

  Pops
  calculate_populations_per_gpu(const float* coords
                              , std::size_t n_rows
                              , std::size_t n_cols
                              , std::vector<float> radii
                              , std::size_t i_from
                              , std::size_t i_to
                              , int i_gpu) {
    using Clustering::Tools::min_multiplicator;
    ASSUME_ALIGNED(coords);
    unsigned int n_radii = radii.size();
    std::vector<float> rad2(n_radii);
    for (std::size_t i=0; i < n_radii; ++i) {
      rad2[i] = radii[i]*radii[i];
    }
    // GPU setup
    hipSetDevice(i_gpu);
    float* d_coords;
    float* d_rad2;
    unsigned int* d_pops;
    hipMalloc((void**) &d_coords
             , sizeof(float) * n_rows * n_cols);
    hipMalloc((void**) &d_pops
             , sizeof(unsigned int) * n_rows * n_radii);
    hipMalloc((void**) &d_rad2
             , sizeof(float) * n_radii);
    check_error("pop-calc device mallocs");
    hipMemset(d_pops
             , 0
             , sizeof(unsigned int) * n_rows * n_radii);
    check_error("pop-calc memset");
    hipMemcpy(d_coords
             , coords
             , sizeof(float) * n_rows * n_cols
             , hipMemcpyHostToDevice);
    hipMemcpy(d_rad2
             , rad2.data()
             , sizeof(float) * n_radii
             , hipMemcpyHostToDevice);
    check_error("pop-calc mem copies");
    int max_shared_mem;
    hipDeviceGetAttribute(&max_shared_mem
                         , hipDeviceAttributeMaxSharedMemoryPerBlock
                         , i_gpu);
    check_error("getting # of GPUs");
    unsigned int block_size = BSIZE_POPS;
    unsigned int shared_mem = 2 * block_size * n_cols * sizeof(float);
    if (shared_mem > max_shared_mem) {
      std::cerr << "error: max. shared mem per block too small on this GPU.\n"
                << "       either reduce BSIZE_POPS or get a better GPU."
                << std::endl;
      exit(EXIT_FAILURE);
    }
    unsigned int block_rng = min_multiplicator(i_to-i_from, block_size);
    Clustering::logger(std::cout) << "# blocks needed: "
                                  << block_rng << std::endl;
    for (unsigned int i=0; i*block_size < n_rows; ++i) {
      population_count <<< block_rng
                         , block_size
                         , shared_mem >>> (i*block_size
                                         , d_coords
                                         , n_rows
                                         , n_cols
                                         , d_rad2
                                         , n_radii
                                         , d_pops
                                         , i_from
                                         , i_to);
    }
    hipDeviceSynchronize();
    check_error("after kernel loop");
    // get partial results from GPU
    std::vector<unsigned int> partial_pops(n_rows*n_radii);
    hipMemcpy(partial_pops.data()
             , d_pops
             , sizeof(unsigned int) * n_rows * n_radii
             , hipMemcpyDeviceToHost);
    // sort into resulting pops
    Pops pops;
    for (unsigned int r=0; r < n_radii; ++r) {
      pops[radii[r]].resize(n_rows, 0);
      for (unsigned int i=i_from; i < i_to; ++i) {
        pops[radii[r]][i] = partial_pops[r*n_rows+i];
      }
    }
    hipFree(d_coords);
    hipFree(d_rad2);
    hipFree(d_pops);
    return pops;
  }

  Pops
  calculate_populations(const float* coords
                      , const std::size_t n_rows
                      , const std::size_t n_cols
                      , std::vector<float> radii) {
    using Clustering::Tools::dim1_sorted_coords;
    using Clustering::Tools::boxlimits;
    ASSUME_ALIGNED(coords);
    std::sort(radii.begin(), radii.end(), std::greater<float>());
    int n_gpus = get_num_gpus();
    int gpu_range = n_rows / n_gpus;
    int i;
    std::vector<Pops> partial_pops(n_gpus);
    #pragma omp parallel for default(none)\
      private(i)\
      firstprivate(n_gpus,n_rows,n_cols,gpu_range)\
      shared(partial_pops,radii,coords)\
      num_threads(n_gpus)\
      schedule(dynamic,1)
    for (i=0; i < n_gpus; ++i) {
      // compute partial populations in parallel
      // on all available GPUs
      partial_pops[i] = calculate_populations_per_gpu(coords
                                                    , n_rows
                                                    , n_cols
                                                    , radii
                                                    , i*gpu_range
                                                    , i == (n_gpus-1)
                                                        ? n_rows
                                                        : (i+1)*gpu_range
                                                    , i);
    }
    Pops pops;
    // combine pops
    for (float r: radii) {
      pops[r].resize(n_rows, 0);
      for (i=0; i < n_rows; ++i) {
        for (unsigned int i_gpu=0; i_gpu < n_gpus; ++i_gpu) {
          pops[r][i] += partial_pops[i_gpu][r][i];
        }
      }
    }
    return pops;
  }

  std::tuple<Neighborhood, Neighborhood>
  nearest_neighbors_per_gpu(const float* coords
                          , const std::size_t n_rows
                          , const std::size_t n_cols
                          , const std::vector<float>& free_energy
                          , std::size_t i_from
                          , std::size_t i_to
                          , int i_gpu) {
    using Clustering::Tools::min_multiplicator;
    ASSUME_ALIGNED(coords);
    // GPU setup
    hipSetDevice(i_gpu);
    hipStream_t streams[N_STREAMS_NH];
    float* d_coords;
    float* d_fe;
    float* d_nh[N_STREAMS_NH];
    float* d_nhhd[N_STREAMS_NH];
    hipMalloc((void**) &d_coords
             , sizeof(float) * n_rows * n_cols);
    hipMalloc((void**) &d_fe
             , sizeof(float) * n_rows);
    for (unsigned int i=0; i < N_STREAMS_NH; ++i) {
      hipMalloc((void**) &d_nh[i]
               , sizeof(float) * n_rows * 2);
      hipMalloc((void**) &d_nhhd[i]
               , sizeof(float) * n_rows * 2);
      hipMemset(d_nh[i]
               , 0
               , sizeof(float) * n_rows * 2);
      hipMemset(d_nhhd[i]
               , 0
               , sizeof(float) * n_rows * 2);
      hipStreamCreate(&streams[i]);
    }
    hipMemcpy(d_coords
             , coords
             , sizeof(float) * n_rows * n_cols
             , hipMemcpyHostToDevice);
    hipMemcpy(d_fe
             , free_energy.data()
             , sizeof(float) * n_rows
             , hipMemcpyHostToDevice);
    int max_shared_mem;
    hipDeviceGetAttribute(&max_shared_mem
                         , hipDeviceAttributeMaxSharedMemoryPerBlock
                         , i_gpu);
    check_error("retrieving max shared mem");
    unsigned int block_size = BSIZE_NH;
    unsigned int shared_mem = 2 * block_size * n_cols * sizeof(float);
    if (shared_mem > max_shared_mem) {
      std::cerr << "error: max. shared mem per block too small on this GPU.\n"
                << "       either reduce block_size for NN search or get a "
                <<        "better GPU." << std::endl;
      exit(EXIT_FAILURE);
    }
    unsigned int block_rng = min_multiplicator(i_to-i_from, block_size);
    for (unsigned int i=0; i*block_size < n_rows; ++i) {
      unsigned int i_stream = i % N_STREAMS_NH;
      nearest_neighbor_search <<< block_rng
                                , block_size
                                , shared_mem
                                , streams[i_stream] >>> (i*block_size
                                                       , d_coords
                                                       , n_rows
                                                       , n_cols
                                                       , d_fe
                                                       , d_nh[i_stream]
                                                       , d_nhhd[i_stream]
                                                       , i_from
                                                       , i_to);
    }
    hipDeviceSynchronize();
    check_error("after kernel loop");
    // initialize neighborhoods
    Neighborhood nh;
    Neighborhood nhhd;
    for (unsigned int i=0; i < n_rows; ++i) {
      nh[i] = {i, std::numeric_limits<float>::max()};
      nhhd[i] = {i, std::numeric_limits<float>::max()};
    }
    // collect partial results from streams
    for (unsigned int i_stream=0; i_stream < N_STREAMS_NH; ++i_stream) {
      std::vector<float> dist_ndx(n_rows * 2);
      auto update_nh = [&dist_ndx,n_rows] (Neighborhood& _nh) -> void {
        for (unsigned int i=0; i < n_rows; ++i) {
          if (dist_ndx[i] < _nh[i].second && dist_ndx[i] != 0) {
            _nh[i] = {(unsigned int) dist_ndx[n_rows+i]
                    , dist_ndx[i]};
          }
        }
      };
      hipMemcpy(dist_ndx.data()
               , d_nh[i_stream]
               , sizeof(float) * n_rows * 2
               , hipMemcpyDeviceToHost);
      update_nh(nh);
      hipMemcpy(dist_ndx.data()
               , d_nhhd[i_stream]
               , sizeof(float) * n_rows * 2
               , hipMemcpyDeviceToHost);
      update_nh(nhhd);
    }
    // device cleanup
    hipFree(d_coords);
    hipFree(d_fe);
    for (unsigned int i=0; i < N_STREAMS_NH; ++i) {
      hipFree(d_nh[i]);
      hipFree(d_nhhd[i]);
    }
    return std::make_tuple(nh, nhhd);
  }

  std::tuple<Neighborhood, Neighborhood>
  nearest_neighbors(const float* coords
                  , const std::size_t n_rows
                  , const std::size_t n_cols
                  , const std::vector<float>& free_energy) {
    int n_gpus = get_num_gpus();
    std::vector<std::tuple<Neighborhood, Neighborhood>> partials(n_gpus);
    unsigned int gpu_range = n_rows / n_gpus;
    unsigned int i_gpu;
    #pragma omp parallel for default(none)\
      private(i_gpu)\
      firstprivate(n_gpus,n_rows,n_cols,gpu_range)\
      shared(partials,coords,free_energy)\
      num_threads(n_gpus)
    for (i_gpu=0; i_gpu < n_gpus; ++i_gpu) {
      partials[i_gpu] = nearest_neighbors_per_gpu(coords
                                                , n_rows
                                                , n_cols
                                                , free_energy
                                                , i_gpu*gpu_range
                                                , i_gpu == (n_gpus-1)
                                                        ? n_rows
                                                        : (i_gpu+1)*gpu_range
                                                , i_gpu);
    }
    // combine partial neighborhood results from different gpus
    Neighborhood nh;
    Neighborhood nhhd;
    std::tie(nh, nhhd) = partials[0];
    for (i_gpu=1; i_gpu < n_gpus; ++i_gpu) {
      Neighborhood partial_nh;
      Neighborhood partial_nhhd;
      std::tie(partial_nh, partial_nhhd) = partials[i_gpu];
      for (unsigned int i=0; i < n_rows; ++i) {
        if (partial_nh[i].second < nh[i].second) {
          nh[i] = partial_nh[i];
        }
        if (partial_nhhd[i].second < nhhd[i].second) {
          nhhd[i] = partial_nhhd[i];
        }
      }
    }
    return std::make_tuple(nh, nhhd);
  }



  std::set<std::size_t>
  high_density_neighborhood(std::vector<float*> d_coords
                          , const std::size_t n_rows
                          , const std::size_t n_cols
                          , std::vector<float*> d_fe
                          , std::vector<int*> d_local_nh
                          , const float free_energy_threshold
                          , std::size_t i_ref
                          , const float max_dist2
                          , int n_gpus) {
    //TODO: finish
  }


  std::vector<std::size_t>
  initial_density_clustering(const std::vector<float>& free_energy
                           , const Neighborhood& nh
                           , const float free_energy_threshold
                           , const float* coords
                           , const std::size_t n_rows
                           , const std::size_t n_cols
                           , const std::vector<std::size_t> initial_clusters) {
    std::vector<std::size_t> clustering;
    std::size_t first_frame_above_threshold;
    double sigma2;
    std::set<std::size_t> visited_frames;
    std::size_t distinct_name;
    // data preparation
    std::tie(clustering
           , first_frame_above_threshold
           , sigma2
           , std::ignore
           , visited_frames
           , distinct_name) = prepare_initial_clustering(free_energy
                                                       , nh
                                                       , free_energy_threshold
                                                       , n_rows
                                                       , initial_clusters);
    // write log
    screening_log(sigma2
                , first_frame_above_threshold
                , fe_sorted);
    // prepare CUDA environment
    int n_gpus = get_num_gpus();
    std::vector<float*> d_coords(n_gpus);
    std::vector<float*> d_fe(n_gpus);
    std::vector<int*> d_local_nh(n_gpus);
    for (int i_gpu=0; i_gpu < n_gpus; ++i_gpu) {
      hipMalloc((void**) &d_coords[i_gpu]
               , sizeof(float) * n_rows * n_cols);
      hipMalloc((void**) &d_fe[i_gpu]
               , sizeof(float) * n_rows);
      hipMalloc((void**) &d_local_nh[i_gpu]
               , sizeof(int) * n_rows);
      hipMemcpy(d_coords[i_gpu]
               , coords
               , sizeof(float) * n_rows * n_cols
               , hipMemcpyHostToDevice);
      hipMemcpy(d_fe[i_gpu]
               , free_energy.data()
               , sizeof(float) * n_rows
               , hipMemcpyHostToDevice);
    }
    // indices inside this loop are in order of sorted(!) free energies
    bool neighboring_clusters_merged = false;
    std::set<std::size_t> local_nh;
    while ( ! neighboring_clusters_merged) {
      neighboring_clusters_merged = true;
      logger(std::cout) << "initial merge iteration" << std::endl;
      for (std::size_t i=0; i < first_frame_above_threshold; ++i) {
        if (visited_frames.count(i) == 0) {
          visited_frames.insert(i);
          // all frames/clusters in local neighborhood should be merged ...
          local_nh = high_density_neighborhood(d_coords
                                             , n_rows
                                             , n_cols
                                             , d_fe
                                             , d_local_nh
                                             , free_energy_threshold
                                             , i
                                             , 4*sigma2
                                             , n_gpus);
          //TODO: profiling!
          //      if this needs lots of time: use OMP parallel sections with
          //      pre-calculated local_nh.
          neighboring_clusters_merged = lump_initial_clusters(local_nh
                                                            , distinct_name
                                                            , clustering
                                                            // CUDA-based high-dens nh
                                                            // uses unsorted indices
                                                            , {}
                                                            , first_frame_above_threshold)
                                     && neighboring_clusters_merged;
        }
      }
    }
    // cleanup CUDA environment
    for (int i_gpu=0; i_gpu < n_gpus; ++i_gpu) {
      hipFree(d_coords[i_gpu]);
      hipFree(d_fe[i_gpu]);
      hipFree(d_local_nh[i_gpu]);
    }
    return normalized_cluster_names(first_frame_above_threshold
                                  , clustering
                                  , fe_sorted);
  }

}}} // end Clustering::Density::CUDA

